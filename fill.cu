//created by xianglizhi, code follows exactly tensorflow fill example which is below
//compile cmd is nvcc -std=c++11 -O3 -I /usr/local/cuda/include -L /usr/local/cuda/lib64 -L /usr/local/lib fill.cu -o fill
//running ./fill
/*
 * .html# Output tensor has shape [2, 3].
    fill([2, 3], 9) ==> [[9, 9, 9]
                     [9, 9, 9]]

 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
template<class T>
__global__ void fill(T *input, T value, unsigned int linear_input_shape){
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(global_id >= linear_input_shape){
        return;
    }
    input[global_id] = value;
}
int main(int argc, char *argv[]){
    float *float_matrix;//declared output buffer
    int *int_matrix;//declared output buffer
    // assuming desired matrix shape is 3,4,5, now we malloc the needed buffer in GPU
    hipMalloc(&float_matrix,3*4*5*sizeof(float));
    hipMalloc(&int_matrix,3*4*5*sizeof(int));
    //we fix block size to 64, feel free to change
    dim3 block(64);
    //how many blocks we need for this job, for here we need (3*4*5 - 1)/64 + 1 blocks
    dim3 grid((3*4*5 - 1)/64 + 1);
    //fill<float><<<grid,block>>>(float_matrx,1.0f,3*4*5);
    fill<int><<<grid,block>>>(int_matrix,1,3*4*5);
    int *host_buffer = (int *)malloc(3*4*5*sizeof(int));
    hipMemcpy(host_buffer,int_matrix,3*4*5*sizeof(int),hipMemcpyDeviceToHost);
    for(unsigned int i = 0; i < 3*4*5; ++i){
        std::cout<<host_buffer[i]<<" ";
    }
    std::cout<<std::endl;
    return 0;
}